#include "hip/hip_runtime.h"
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include <stb/stb_image_write.h>

#include <iostream>
#include <hip/hip_runtime.h>

#define CUDA_CHECK(val) check_cuda((val), #val, __FILE__, __LINE__)
void check_cuda(hipError_t result, char const *const func, const char *const file, int const line)
{
    if (result)
    {
        std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " << file << ":" << line << " '" << func << "' \n";
        // Make sure we call CUDA Device Reset before exiting
        hipDeviceReset();
        exit(99);
    }
}

__global__ void render(float *fb, int max_x, int max_y)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if ((i >= max_x) || (j >= max_y))
        return;
    int pixel_index = j * max_x * 3 + i * 3;
    fb[pixel_index + 0] = float(i) / max_x;
    fb[pixel_index + 1] = float(j) / max_y;
    fb[pixel_index + 2] = 0.2;
}

int main()
{
    int w = 1280;
    int h = 720;

    int tx = 8;
    int ty = 8;

    std::cout << "Rendering a " << w << "x" << h << " image ";
    std::cout << "in " << tx << "x" << ty << " blocks.\n";

    float *fb;
    checkCudaErrors(hipMallocManaged((void **)&fb, fb_size));
    int num_px = w * h;
    // Render our buffer
    dim3 blocks(w/tx+1,h/ty+1);
    dim3 threads(tx,ty);
    render<<<blocks, threads>>>(fb, nx, ny);
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());

    


    return 0;
}